#include "hip/hip_runtime.h"
#include <cstdlib>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <iostream>
#include <sstream>
#include <string>
#include <chrono>
#include "check.hpp"

using namespace std::chrono;

int main(int argc, char const *argv[])
{
    // input setting
    int M,N,K;
    std::cout<<"input M,N,K: ";
    std::cin>>M>>N>>K;
    std::cout<<std::endl;
    
    // host's space
    double *A,*B,*C,alpha=1.0,beta=0.0;
    A=new double[M*N];
    B=new double[N*K];
    C=new double[M*K];

    // initialize
    srand(time(NULL));
    int tmp=M*N;
    for (int i=0;i<tmp;i++) A[i]=(double)(rand()%6400)/100;
    tmp=N*K;
    for (int i=0;i<tmp;i++) B[i]=(double)(rand()%6400)/100;
    bzero(C,M*K*sizeof(double));

    for (int i=0;i<M;i++)
    {
        for (int j=0;j<N;j++)
        {
            std::cout<<A[i*N+j]<<' ';
        }
        std::cout<<std::endl;
    }
    std::cout<<"=================================="<<std::endl;
    for (int i=0;i<N;i++)
    {
        for (int j=0;j<K;j++)
        {
            std::cout<<B[i*K+j]<<' ';
        }
        std::cout<<std::endl;
    }
    std::cout<<"=================================="<<std::endl;
    
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double *device_A, *device_B, *device_C;
    // device's space
    hipMalloc((void **)&device_A,sizeof(double)*M*N);
    hipMalloc((void **)&device_B,sizeof(double)*N*K);
    hipMalloc((void **)&device_C,sizeof(double)*M*K);
    // pass value

    const double *device_A_const = device_A;
    const double *device_B_const = device_B;

    auto start=system_clock::now();
    hipMemcpy(device_A,A,sizeof(double)*M*N,hipMemcpyHostToDevice);
    hipMemcpy(device_B,B,sizeof(double)*N*K,hipMemcpyHostToDevice);
    hipblasDgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,K,M,N,&alpha,
        device_B_const,K,
        device_A_const,N,&beta,
        device_C,K);
    hipMemcpy(C,device_C,M*K*sizeof(double),hipMemcpyDeviceToHost);
    auto end=system_clock::now();
    auto duration=duration_cast<microseconds>(end-start);
    std::stringstream ss;
    ss<<"echo '";
    ss<<"M="<<M<<",N="<<N<<",K="<<K<<"\t";
    ss<<"time="<<std::fixed<<double(duration.count())<<"us";
    ss<<"' >> BLAS_GEMM.log";
    std::string cmd=ss.str();
    std::cout<<cmd<<std::endl;
    system(cmd.c_str());

    for (int i=0;i<M;i++)
    {
        for (int j=0;j<K;j++)
        {
            std::cout<<C[i*K+j]<<' ';
        }
        std::cout<<std::endl;
    }

    checkMatrix(A,B,C,M,N,K);
    
    hipblasDestroy(handle);
    return 0;
}

