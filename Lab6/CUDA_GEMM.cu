#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <string>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "check.hpp"

using namespace std::chrono;

__global__ 
void mat_mul(double *A,double *B,double *C,int *p_M,int *p_N,int *p_K)
{
    //__syncthreads();
    int M,N,K;
    M=*p_M;
    N=*p_N;
    K=*p_K;
    // get self idx
    int i=blockIdx.x*blockDim.x+threadIdx.x; 
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if (i>=M||j>=K) return;
    

    // cal
    C[i*K+j]=0.0;
    for (int n=0;n<N;n++) C[i*K+j]+=A[i*N+n]*B[n*K+j];
}

int main(int argc, char *argv[])
{
    double *A,*B,*C;
    double *device_A,*device_B,*device_C;

    // input settings
    int blockNum;
    std::cout<<"input block number: ";
    std::cin>>blockNum;
    std::cout<<std::endl;

    int M,N,K;
    std::cout<<"input M,N,K: ";
    std::cin>>M>>N>>K;
    std::cout<<std::endl;
    
    // host's space
    A=new double[M*N];
    B=new double[N*K];
    C=new double[M*K];

    // initialize
    srand(time(NULL));
    int tmp=M*N;
    for (int i=0;i<tmp;i++) A[i]=(double)(rand()%6400)/100;
    tmp=N*K;
    for (int i=0;i<tmp;i++) B[i]=(double)(rand()%6400)/100;
    bzero(C,M*K*sizeof(double));

    for (int i=0;i<M;i++)
    {
        for (int j=0;j<N;j++)
        {
            std::cout<<A[i*N+j]<<' ';
        }
        std::cout<<std::endl;
    }
    std::cout<<"=================================="<<std::endl;
    for (int i=0;i<N;i++)
    {
        for (int j=0;j<K;j++)
        {
            std::cout<<B[i*K+j]<<' ';
        }
        std::cout<<std::endl;
    }
    std::cout<<"=================================="<<std::endl;

    // device's space
    hipMalloc((void **)&device_A,sizeof(double)*M*N);
    hipMalloc((void **)&device_B,sizeof(double)*N*K);
    hipMalloc((void **)&device_C,sizeof(double)*M*K);
    int *device_M,*device_N,*device_K;
    hipMalloc((void **)&device_M,sizeof(int));
    hipMalloc((void **)&device_N,sizeof(int));
    hipMalloc((void **)&device_K,sizeof(int));

    
    auto start=system_clock::now();
    // pass value
    hipMemcpy(device_A,A,sizeof(double)*M*N,hipMemcpyHostToDevice);
    hipMemcpy(device_B,B,sizeof(double)*N*K,hipMemcpyHostToDevice);
    hipMemcpy(device_M,&M,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(device_N,&N,sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(device_K,&K,sizeof(int),hipMemcpyHostToDevice);

    dim3 grid(M/blockNum+1,K);
    dim3 block(blockNum);
    mat_mul<<<grid,block>>>(device_A,device_B,device_C,device_M,device_N,device_K);

    hipMemcpy(C,device_C,sizeof(double)*M*K,hipMemcpyDeviceToHost);

    auto end=system_clock::now();
    auto duration=duration_cast<microseconds>(end-start);
    std::stringstream ss;
    ss<<"echo '";
    ss<<"M="<<M<<",N="<<N<<",K="<<K<<"\t";
    ss<<"blocks="<<blockNum<<",time="<<std::fixed<<double(duration.count())<<"us";
    ss<<"' >> CUDA_GEMM.log";
    std::string cmd=ss.str();
    std::cout<<cmd<<std::endl;
    system(cmd.c_str());

    for (int i=0;i<M;i++)
    {
        for (int j=0;j<K;j++)
        {
            std::cout<<C[i*K+j]<<' ';
        }
        std::cout<<std::endl;
    }

    checkMatrix(A,B,C,M,N,K);

    delete[] A;
    delete[] B;
    delete[] C;

}