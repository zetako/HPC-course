#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <string>
#include <string.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <chrono>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "check.hpp"
#include <omp.h>

using namespace std::chrono;

__global__ 
void mat_mul(
    double *A,double *B,double *C,
    int *p_M,int *p_N,int *p_K,
    int *p_offset)
{
    //__syncthreads();
    int M,N,K;
    M=*p_M;
    N=*p_N;
    K=*p_K;
    int offset=*p_offset;
    // get self idx
    int i=blockIdx.x*blockDim.x+threadIdx.x; 
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    if ((i+offset)>=M||j>=K) return;

    // cal
    C[i*K+j]=0.0;
    for (int n=0;n<N;n++) C[i*K+j]+=A[i*N+n]*B[n*K+j];
}

int main(int argc, char *argv[])
{
    double *A,*B,*C;
    double *device_A,*device_B,*device_C;

    // input settings
    int threadNum;
    std::cout<<"input thread number: ";
    std::cin>>threadNum;
    std::cout<<std::endl;
    switch (threadNum)
    {
        case 1:
        case 2:
        case 4:
        case 8:
            break;
        default:
            std::cout<<"invaild thread number!"<<std::endl;
            return -1;
            break;
    }

    int blockNum;
    std::cout<<"input block number(pre thread): ";
    std::cin>>blockNum;
    std::cout<<std::endl;

    int M,N,K;
    std::cout<<"input M,N,K: ";
    std::cin>>M>>N>>K;
    std::cout<<std::endl;
    
    // host's space
    A=new double[M*N];
    B=new double[N*K];
    C=new double[M*K];

    // initialize
    srand(time(NULL));
    int tmp=M*N;
    for (int i=0;i<tmp;i++) A[i]=(double)(rand()%6400)/100;
    tmp=N*K;
    for (int i=0;i<tmp;i++) B[i]=(double)(rand()%6400)/100;
    bzero(C,M*K*sizeof(double));

    for (int i=0;i<M;i++)
    {
        for (int j=0;j<N;j++)
        {
            std::cout<<A[i*N+j]<<' ';
        }
        std::cout<<std::endl;
    }
    std::cout<<"=================================="<<std::endl;
    for (int i=0;i<N;i++)
    {
        for (int j=0;j<K;j++)
        {
            std::cout<<B[i*K+j]<<' ';
        }
        std::cout<<std::endl;
    }
    std::cout<<"=================================="<<std::endl;

    
    
    auto start=system_clock::now();
    #pragma omp parallel for num_threads(threadNum)
    for (int i=0;i<threadNum;i++)
    {
        // device's space
        hipMalloc((void **)&device_A,sizeof(double)*M*N/threadNum);
        hipMalloc((void **)&device_B,sizeof(double)*N*K);
        hipMalloc((void **)&device_C,sizeof(double)*M*K/threadNum);
        int *device_M,*device_N,*device_K;
        hipMalloc((void **)&device_M,sizeof(int));
        hipMalloc((void **)&device_N,sizeof(int));
        hipMalloc((void **)&device_K,sizeof(int));
        int *device_offset;
        hipMalloc((void **)&device_offset,sizeof(int));

        // decide offset
        double *offset_A,*offset_C;
        offset_A=A+((M*N)/threadNum)*i;
        offset_C=C+((M*K)/threadNum)*i;

        // pass value
        hipMemcpy(device_A,offset_A,sizeof(double)*M*N/threadNum,hipMemcpyHostToDevice);
        hipMemcpy(device_B,B,sizeof(double)*N*K,hipMemcpyHostToDevice);
        hipMemcpy(device_M,&M,sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(device_N,&N,sizeof(int),hipMemcpyHostToDevice);
        hipMemcpy(device_K,&K,sizeof(int),hipMemcpyHostToDevice);
        int offset=M/threadNum*i;
        hipMemcpy(device_offset,&offset,sizeof(int),hipMemcpyHostToDevice);
    
        dim3 grid((M/blockNum/threadNum)+1,K);
        dim3 block(blockNum);
        mat_mul<<<grid,block>>>(device_A,device_B,device_C,device_M,device_N,device_K,device_offset);
    
        hipMemcpy(offset_C,device_C,sizeof(double)*M*K/threadNum,hipMemcpyDeviceToHost);
    }

    auto end=system_clock::now();
    auto duration=duration_cast<microseconds>(end-start);
    std::stringstream ss;
    ss<<"echo '";
    ss<<"M="<<M<<",N="<<N<<",K="<<K<<"\t";
    ss<<"threads="<<threadNum<<",blocks="<<blockNum<<'\t';
    ss<<"time="<<std::fixed<<double(duration.count())<<"us";
    ss<<"' >> omp_CUDA_GEMM.log";
    std::string cmd=ss.str();
    std::cout<<cmd<<std::endl;
    system(cmd.c_str());

    for (int i=0;i<M;i++)
    {
        for (int j=0;j<K;j++)
        {
            std::cout<<C[i*K+j]<<' ';
        }
        std::cout<<std::endl;
    }

    checkMatrix(A,B,C,M,N,K);

    delete[] A;
    delete[] B;
    delete[] C;

    return 0;

}