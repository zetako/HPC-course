#include <hip/hip_runtime.h>
/*
    status:☹️
    sequence:NCHW
*/

struct arg
{
    int image_height;
    int image_width;
    int channel;
    int stride;
    int filter_height;
    int filter_width;
};

__global__ void conv_2d(float *image,float *filter,float *out,arg *args)
{
    int x=blockIdx.x*blockDim.x+threadIdx.x; 
    int y=blockIdx.y*blockDim.y+threadIdx.y;

    int out_height=args->image_height-args->filter_height;
    int out_width=args->image_width-args->filter_width;

    float local=0.0;
    for (int i=0;i<args->filter_height;i++)
    {
        for (int j=0;j<args->filter_width;i++)
            local+=filter[i*args->filter_width+j]*image[(i+x)*args->image_width+(j+y)];
    }
}